#include "hip/hip_runtime.h"
#include "../include/utils.h"

using namespace std;









hipEvent_t start, stop;

void checkCUDAError (const char *msg) {
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		cerr << "Cuda error: " << msg << ", " << hipGetErrorString( err) << endl;
		exit(-1);
	}
}

// These are specific to measure the execution of only the kernel execution - might be useful
void startKernelTime (void) {
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
}

void stopKernelTime (void) {
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << endl << "Basic profiling: " << milliseconds << " ms have elapsed for the kernel execution" << endl << endl;
}


















// function that generates random float values for N points
// and assigns the first K as centroids
void init(int N, int K, float *px, float *py, float *cx, float *cy){
	srand(10);
	for(int i = 0; i < N; i++){
		px[i] = (float) rand() / RAND_MAX;
		py[i] = (float) rand() / RAND_MAX;
	}

	for(int i = 0; i < K; i++){
		cx[i] = px[i];
		cy[i] = py[i];
	}
}

// finds the most appropriate cluster for a given point based on euclidean distance
inline int findCluster(int K, float px, float py, float *cx, float *cy){
	float dist_min = 2; 
	int min = -1;

	for(int j = 0; j < K; j++){
		float dist = (cx[j] - px) * (cx[j] - px) + (cy[j] - py) * (cy[j] - py);

		if (dist < dist_min){
			dist_min = dist;
			min = j;
		}
	}

	return min;
}

// associates points to a cluster 
int attributeClusters(int N, int K, int THREADS, float *px, float *py, 
			float *cx, float *cy, int *point_cluster){
	int changed = 0;
	int cluster;
	for(int i = 0; i < N; i++){
		cluster = findCluster(K, px[i], py[i], cx, cy);
		if (cluster != point_cluster[i]){
			changed = 1;
			point_cluster[i] = cluster;
		}
	}

	return changed;
}

// calculates the centroids of each cluster
void rearrangeCluster(int N, int K, int THREADS, float *px, float *py, 
				float *cx, float *cy, int *point_cluster, int *size){
	// size keeps track of how much points are in each cluster
	/* x and y contain the sum of x and y values (respectively) 
	   of the points that belong to the cluster */
	float x[K], y[K];

	for(int i = 0; i < K; i++){
		size[i] = 0;
		x[i] = 0;
		y[i] = 0;
	}

	for(int i = 0; i < N; i++){
		size[point_cluster[i]]++;
		x[point_cluster[i]] += px[i];
		y[point_cluster[i]] += py[i];
	}

	for(int i = 0; i < K; i++){
		cx[i] = x[i] / size[i];
		cy[i] = y[i] / size[i];
	}
}

__global__ 
void attributeCluster(float *cx, float *cy, float *px, float *py, 
			int *point_cluster, int *changed, const int N, const int K){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < N){
		float x = px[id];
		float y = py[id];
		
		//int changed = 0;
		int cluster = -1;
		float dist_min = 2; 

		for(int j = 0; j < K; j++){
			float dist = (cx[j] - x) * (cx[j] - x) + (cy[j] - y) * (cy[j] - y);

			if (dist < dist_min){
				dist_min = dist;
				cluster = j;
			}
		}


		if (cluster != point_cluster[id]){
			point_cluster[id] = cluster;
			if (*changed == 0)
				atomicAdd(changed, 1);
		}
	}
} 

int clustersChanged(int blocks, int *changed){
	int res = 0;

	for(int i = 0; i < blocks; i++){
		if (changed[i] != 0){
			res = 1;
			break;
		}
	}

	return res;
}

// executes k-means algorithm and returns how many iterations were made
void kmeans(int N, int K, int THREADS, float *px, float *py, 
		float *cx, float *cy, int *point_cluster, int *size){

	int blocks = N/THREADS + 1; 
	float *dcx, *dcy, *dpx, *dpy;
	int *dpoint_cluster, *dchanged, changed[1];
	memset(point_cluster, 0, N * sizeof(int));

	hipMalloc((void**) &dcx, K * sizeof(float));
	hipMalloc((void**) &dcy, K * sizeof(float));
	hipMalloc((void**) &dpx, N * sizeof(float));
	hipMalloc((void**) &dpy, N * sizeof(float));
	hipMalloc((void**) &dpoint_cluster, N * sizeof(int));
	hipMalloc((void**) &dchanged, sizeof(int));
	checkCUDAError("malloc");

	hipMemcpy(dcx, cx, K * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dcy, cy, K * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dpx, px, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dpy, py, N * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError("cpy");

	int i;
	for(i = 0; i < 20; i++){
		hipMemcpy(dpoint_cluster, point_cluster, N * sizeof(int), hipMemcpyHostToDevice);
		hipMemset(dchanged, 0, sizeof(int));
		//startKernelTime ();
		attributeCluster <<< blocks, THREADS >>> (dcx, dcy, dpx, dpy, dpoint_cluster, dchanged, N, K);
		//stopKernelTime ();

		hipMemcpy(changed, dchanged, sizeof(int), hipMemcpyDeviceToHost);
		if (*changed == 0)
			break;

		hipMemcpy(point_cluster, dpoint_cluster, N * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(cx, dcx, K * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(cy, dcy, K * sizeof(float), hipMemcpyDeviceToHost);
		rearrangeCluster(N, K, THREADS, px, py, cx, cy, point_cluster, size);

		hipMemcpy(dcx, cx, K * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dcy, cy, K * sizeof(int), hipMemcpyHostToDevice);
	}

	printInfo(N, K, cx, cy, size, i);
}

// prints information about the clusters
void printInfo(int N, int K, float *cx, float *cy, int *size, int iterations){
	printf("N = %d, K = %d\n", N, K);

	for(int i = 0; i < K; i++)
		printf("[%d]: (%.3f, %.3f) : Size: %d\n", i, cx[i], cy[i], size[i]);

	printf("Iterations: %d\n", iterations);
}
