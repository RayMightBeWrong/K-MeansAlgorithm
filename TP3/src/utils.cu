#include "hip/hip_runtime.h"
#include "../include/utils.h"

using namespace std;


// function that generates random float values for N points
// and assigns the first K as centroids
void init(int N, float *px, float *py, float *cx, float *cy){
	srand(10);
	for(int i = 0; i < N; i++){
		px[i] = (float) rand() / RAND_MAX;
		py[i] = (float) rand() / RAND_MAX;
	}

	for(int i = 0; i < K; i++){
		cx[i] = px[i];
		cy[i] = py[i];
	}
}


__global__ 
void attributeCluster(float *cx, float *cy, float *px, float *py, 
			int *point_cluster, int *changed, const int N){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < N){
		__shared__ float lcx[K];
		__shared__ float lcy[K];
		float x = px[id];
		float y = py[id];

		int lid = threadIdx.x; 
		if (lid < K){
			lcx[lid] = cx[lid];
			lcy[lid] = cy[lid];
		}
		__syncthreads();
		
		int cluster = -1;
		float dist_min = 2; 

		for(int j = 0; j < K; j++){
			float dist = (lcx[j] - x) * (lcx[j] - x) + (lcy[j] - y) * (lcy[j] - y);

			if (dist < dist_min){
				dist_min = dist;
				cluster = j;
			}
		}

		if (cluster != point_cluster[id]){
			point_cluster[id] = cluster;
			if (*changed == 0)
				atomicAdd(changed, 1);
		}
	}
} 


__global__
void calcClusterValues(const int N, const int THREADS, const float *px, const float *py, 
			float *x, float *y, int *size, const int *point_cluster){

	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < N){
		int lid = threadIdx.x;
		__shared__ int lsize[K];
		__shared__ float lx[K];
		__shared__ float ly[K];

		if (lid < K){
			lsize[lid] = 0;
			lx[lid] = 0;	
			ly[lid] = 0;
		}
		__syncthreads();

		int cluster = point_cluster[id];
		atomicAdd(&lx[cluster], px[id]);
		atomicAdd(&ly[cluster], py[id]);
		atomicAdd(&lsize[cluster], 1);
		__syncthreads();

		if (lid < K){
			atomicAdd(&size[lid], lsize[lid]);
			atomicAdd(&x[lid], lx[lid]);
			atomicAdd(&y[lid], ly[lid]);
		}
		__syncthreads();
	}
}

// calculates the centroids of each cluster
void rearrangeCluster(float *cx, float *cy, float *x, float *y, int *size){
	for(int i = 0; i < K; i++){
		cx[i] = x[i] / size[i];
		cy[i] = y[i] / size[i];
	}
}

// executes k-means algorithm and returns how many iterations were made
void kmeans(const int N, const int THREADS, float *px, float *py, 
		float *cx, float *cy, int *point_cluster){

	int blocks = N/THREADS + 1; 
	float *dcx, *dcy, *dpx, *dpy;
	int *dpoint_cluster, *dchanged, changed[1], *dsize;
	int *size = (int*) malloc(K * sizeof(int));
	float *x = (float*) malloc(K * sizeof(float)), 
	      *y = (float*) malloc(K * sizeof(float));
	memset(point_cluster, 0, N * sizeof(int));

	hipMalloc((void**) &dcx, K * sizeof(float));
	hipMalloc((void**) &dcy, K * sizeof(float));
	hipMalloc((void**) &dsize, K * sizeof(int));
	hipMalloc((void**) &dpx, N * sizeof(float));
	hipMalloc((void**) &dpy, N * sizeof(float));
	hipMalloc((void**) &dpoint_cluster, N * sizeof(int));
	hipMalloc((void**) &dchanged, sizeof(int));

	hipMemcpy(dpx, px, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dpy, py, N * sizeof(float), hipMemcpyHostToDevice);

	int i;
	for(i = 0; i < 20; i++){
		hipMemcpy(dcx, cx, K * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dcy, cy, K * sizeof(int), hipMemcpyHostToDevice);
		hipMemset(dchanged, 0, sizeof(int));
		attributeCluster <<< blocks, THREADS >>> (dcx, dcy, dpx, dpy, dpoint_cluster, dchanged, N);
		hipDeviceSynchronize();

		hipMemcpy(changed, dchanged, sizeof(int), hipMemcpyDeviceToHost);
		if (*changed == 0)
			break;

		hipMemset(dsize, 0, K * sizeof(int));
		hipMemset(dcx, 0, K * sizeof(int));
		hipMemset(dcy, 0, K * sizeof(int));
		calcClusterValues <<< blocks, THREADS >>> (N, THREADS, dpx, dpy, dcx, dcy, dsize, dpoint_cluster);
		hipDeviceSynchronize();

		hipMemcpy(x, dcx, K * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(y, dcy, K * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(size, dsize, K * sizeof(float), hipMemcpyDeviceToHost);
		
		rearrangeCluster(cx, cy, x, y, size);
	}
	
	printInfo(N, cx, cy, size, i);
}

// prints information about the clusters
void printInfo(int N, float *cx, float *cy, int *size, int iterations){
	printf("N = %d, K = %d\n", N, K);

	for(int i = 0; i < K; i++)
		printf("[%d]: (%.3f, %.3f) : Size: %d\n", i, cx[i], cy[i], size[i]);

	printf("Iterations: %d\n", iterations);
}
