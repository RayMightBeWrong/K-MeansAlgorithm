#include "../include/utils.h"


hipEvent_t start, stop;

void checkCUDAError (const char *msg) {
        hipError_t err = hipGetLastError();
        if( hipSuccess != err) {
                cerr << "Cuda error: " << msg << ", " << hipGetErrorString( err) << endl;
                exit(-1);
        }
}

// These are specific to measure the execution of only the kernel execution - might be useful
void startKernelTime () {
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
}

void stopKernelTime () {
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        cout << endl << "Basic profiling: " << milliseconds << " ms have elapsed for the kernel execution" << endl << endl;
}



int main(int argc, char **argv){
	// N is the number of points 
	int N = atoi(argv[1]);
	
	// px contains the x values of the points
	// py contains the y values of the points
	float *px = (float*) malloc(sizeof(float) * N);
	float *py = (float*) malloc(sizeof(float) * N);
	
	// cx contains the x values of every cluster's centroids 
	// cy contains the y values of every cluster's centroids 
	float *cx = (float*) malloc(sizeof(float) * K);
	float *cy = (float*) malloc(sizeof(float) * K);

	// keeps track of which cluster a point belongs to
	int *point_cluster = (int*) malloc(sizeof(int) * N);

	// init gives random values to points and clusters
	// algorithm executes the k-means algorithm and prints the information out
	init(N, px, py, cx, cy);

	if (argc == 3){
		int THREADS = atoi(argv[2]);
		kmeans(N, THREADS, px, py, cx, cy, point_cluster);
	}

	return 0;
}
